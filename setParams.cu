#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <iostream>

using namespace std;

const int SIZE = 8;

void init(float *A) {
  for (size_t i = 0; i < SIZE; i++) {
    A[i] = static_cast<float>(i);
  }
}

void print(float *A) {
  for (size_t i = 0; i < SIZE; i++) {
    cout << A[i] << " ";
  }
  cout << "\n\n";
}

__global__ void kernel(float *array) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < SIZE)
    array[index] += 1.f;
}

int main() {
  // Host array
  float *A_h;
  checkCudaErrors(hipHostMalloc(reinterpret_cast<void **>(&A_h), SIZE * sizeof(float)));

  // Device array
  float *A_d;
  hipMalloc(reinterpret_cast<void **>(&A_d), SIZE * sizeof(float));

  init(A_h);
  cout << "Results after init:\n";
  print(A_h);

  hipStream_t stream;
  hipStreamCreate(&stream);

  hipGraph_t graph;
  hipGraphExec_t instance;

  // Create graph
  hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

  hipMemcpyAsync(reinterpret_cast<void *>(A_d), reinterpret_cast<void *>(A_h),
                  SIZE * sizeof(float), hipMemcpyHostToDevice, stream);
  kernel<<<1, SIZE, 0, stream>>>(A_d);
  hipMemcpyAsync(reinterpret_cast<void *>(A_h), reinterpret_cast<void *>(A_d),
                  SIZE * sizeof(float), hipMemcpyDeviceToHost, stream);

  hipStreamEndCapture(stream, &graph);

  checkCudaErrors(hipGraphInstantiate(&instance, graph, NULL, NULL, 0));
  checkCudaErrors(hipGraphLaunch(instance, stream));

  cout << "Results from kernel:\n";
  print(A_h);

  size_t nNodes;
  // Get the number of graph nodes first
  hipGraphGetNodes(graph, NULL, &nNodes);
  cout << "Number of nodes in graph: " << nNodes << "\n";
  // Then get the nodes
  hipGraphNode_t nodes[nNodes];
  hipGraphGetNodes(graph, nodes, &nNodes);

  // Get memcpy node parameters
  // https://docs.nvidia.com/cuda/cuda-runtime-api/structcudaMemcpy3DParms.html#structcudaMemcpy3DParms
  hipMemcpy3DParms params;
  hipGraphMemcpyNodeGetParams(nodes[0], &params);

  // Print the memcpy kind
  // hipMemcpyHostToHost = 0
  // hipMemcpyHostToDevice = 1
  // hipMemcpyDeviceToHost = 2
  // hipMemcpyDeviceToDevice = 3
  // hipMemcpyDefault = 4
  cout << "kind: " << params.kind << "\n";

  cout << "depth: " << params.extent.depth << "\t";
  cout << "height: " << params.extent.height << "\t";
  cout << "width: " << params.extent.width << "\n";
  
  cout << "A_h: " << A_h << "\t";
  cout << params.srcPtr.ptr << "\n";

  cout << "A_d: " << A_d << "\t";
  cout << params.dstPtr.ptr << "\n";

  // hipKernelNodeParams knp;
  // hipGraphKernelNodeGetParams(nodesQ[0], &knp);
  // hipGraphExecKernelNodeSetParams(graphExec, nodesP[0], &knp);
  // hipGraphLaunch(graphExec, stream);
  // hipDeviceSynchronize();

  hipStreamDestroy(stream);

  hipHostFree(A_h);
  hipFree(A_d);

  return 0;
}
