#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <iostream>

const int SIZE = 32;

void init(float *A) {
  for (size_t i = 0; i < SIZE; i++) {
    A[i] = static_cast<float>(i);
  }
}

void print(float *A) {
  for (size_t i = 0; i < SIZE; i++) {
    std::cout << A[i] << " ";
  }
  std::cout << "\n";
}

__global__ void kernel(float *array) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < SIZE)
    array[index] += 1.f;
}

int main() {
  // Host array
  float *A_h;
  checkCudaErrors(hipHostMalloc(reinterpret_cast<void **>(&A_h), SIZE * sizeof(float)));

  // Device array
  float *A_d;
  hipMalloc(reinterpret_cast<void **>(&A_d), SIZE * sizeof(float));

  init(A_h);
  std::cout << "results after init:\n";
  print(A_h);

  hipStream_t stream;
  hipStreamCreate(&stream);

  hipGraph_t graph;
  hipGraphExec_t instance;

  // Create graph
  hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

  hipMemcpyAsync(reinterpret_cast<void *>(A_d), reinterpret_cast<void *>(A_h),
                  SIZE * sizeof(float), hipMemcpyHostToDevice, stream);
  kernel<<<1, 32, 0, stream>>>(A_d);
  hipMemcpyAsync(reinterpret_cast<void *>(A_h), reinterpret_cast<void *>(A_d),
                  SIZE * sizeof(float), hipMemcpyDeviceToHost, stream);

  hipStreamEndCapture(stream, &graph);

  checkCudaErrors(hipGraphInstantiate(&instance, graph, NULL, NULL, 0));
  checkCudaErrors(hipGraphLaunch(instance, stream));

  std::cout << "results from kernel:\n";
  print(A_h);

  // size_t numP, numQ;
  // hipGraphGetNodes(graphP, NULL, &numP);
  // hipGraphGetNodes(graphQ, NULL, &numQ);

  // hipGraphNode_t nodesP[numP], nodesQ[numQ];
  // hipGraphGetNodes(graphP, nodesP, &numP);
  // hipGraphGetNodes(graphQ, nodesQ, &numQ);

  // hipKernelNodeParams knp;
  // hipGraphKernelNodeGetParams(nodesQ[0], &knp);
  // hipGraphExecKernelNodeSetParams(graphExec, nodesP[0], &knp);
  // hipGraphLaunch(graphExec, stream);
  // hipDeviceSynchronize();

  hipStreamDestroy(stream);

  hipHostFree(A_h);
  hipFree(A_d);

  return 0;
}
