#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <iostream>
#include <chrono>

using namespace std;
using namespace std::chrono;

const int SIZE = 8;
const int N = 512 * 512;

void init(float *A, int size) {
  for (int i = 0; i < size; i++) {
    A[i] = static_cast<float>(i);
  }
}

void print(float *A) {
  for (int i = 0; i < SIZE; i++) {
    cout << A[i] << " ";
  }
  cout << "\n\n";
}

__global__ void kernel(float *array, int size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    array[index] += 1.f;
    //if (index == 0)
    //  printf("### array[%d] = %f\tArray size: %d\n", index, array[index], size);
  }
}

void runGraph(float* B_h, float* B_d, int size, hipGraph_t& graph, hipGraphExec_t& instance,
    hipStream_t& stream) {
  hipMemcpy3DParms dParams0, dParams2;
  hipKernelNodeParams kParams;

  size_t nNodes;

  // Get the number of graph nodes first
  hipGraphGetNodes(graph, NULL, &nNodes);
  // Then get nodes
  hipGraphNode_t nodes[nNodes];
  hipGraphGetNodes(graph, nodes, &nNodes);

  dim3 nthreads(size);
  
  high_resolution_clock::time_point start = high_resolution_clock::now();
  for (int i = 0; i < N; i++) {
    auto ptr = &B_h[i*size];

    // #################### Update node 0 ####################
    checkCudaErrors(hipGraphMemcpyNodeGetParams(nodes[0], &dParams0));
    //dParams0.srcArray = NULL;
    //dParams0.srcPos = make_hipPos(0, 0, 0);
    dParams0.srcPtr = make_hipPitchedPtr(ptr, size * sizeof(float), 1, 1);

    //dParams0.dstArray = NULL;
    //dParams0.dstPos = make_hipPos(0, 0, 0);
    dParams0.dstPtr = make_hipPitchedPtr(B_d, size * sizeof(float), 1, 1);
    dParams0.extent = make_hipExtent(size * sizeof(float), 1, 1);
    //dParams0.kind = hipMemcpyHostToDevice;
    checkCudaErrors(hipGraphExecMemcpyNodeSetParams(instance, nodes[0], &dParams0));

    // #################### Update node 1 ####################
    checkCudaErrors(hipGraphKernelNodeGetParams(nodes[1], &kParams));
    //kParams.func = (void *)kernel;
    //kParams.gridDim = nblocks;
    kParams.blockDim = nthreads;
    //kParams.sharedMemBytes = 0;
    void *kernelArgs[2] = {(void *)&B_d, (void *)&size};
    kParams.kernelParams = kernelArgs;
    //kParams.extra = NULL;
    checkCudaErrors(hipGraphExecKernelNodeSetParams(instance, nodes[1], &kParams));

    // #################### Update node 2 ####################
    hipGraphMemcpyNodeGetParams(nodes[2], &dParams2);
    //dParams2.srcArray = NULL;
    //dParams2.srcPos = make_hipPos(0, 0, 0);
    dParams2.srcPtr = make_hipPitchedPtr(B_d, size * sizeof(float), 1, 1);
    //dParams2.dstArray = NULL;
    //dParams2.dstPos = make_hipPos(0, 0, 0);
    dParams2.dstPtr = make_hipPitchedPtr(ptr, size * sizeof(float), 1, 1);
    dParams2.extent = make_hipExtent(size * sizeof(float), 1, 1);
    //dParams2.kind = hipMemcpyDeviceToHost;
    checkCudaErrors(hipGraphExecMemcpyNodeSetParams(instance, nodes[2], &dParams2));

    // Relaunch the graph with new parameters
    checkCudaErrors(hipGraphLaunch(instance, stream));
    checkCudaErrors(hipStreamSynchronize(stream));
  }
  high_resolution_clock::time_point end = high_resolution_clock::now();
  duration<double> time = duration_cast<duration<double>>(end - start);
  cout << "Graph runtime: " << time.count() << "\n";
  //print(&B_h[(N-1)*size]);
}

void runKernels(float* B_h, float* B_d, int size, hipStream_t &stream) {
  dim3 nblocks(1);
  dim3 nthreads(size);

  high_resolution_clock::time_point start = high_resolution_clock::now();
  for (int i = 0; i < N; i++) {
    auto ptr = &B_h[i*size];

    hipMemcpyAsync(reinterpret_cast<void *>(B_d),
                    reinterpret_cast<void *>(ptr), size * sizeof(float),
                    hipMemcpyHostToDevice, stream);
    kernel<<<nblocks, nthreads, 0, stream>>>(B_d, size);
    hipMemcpyAsync(reinterpret_cast<void *>(ptr),
                    reinterpret_cast<void *>(B_d), size * sizeof(float),
                    hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);
  }
  high_resolution_clock::time_point end = high_resolution_clock::now();
  duration<double> time = duration_cast<duration<double>>(end - start);
  cout << "Kernels runtime: " << time.count() << "\n";
  //print(&B_h[(N-1)*size]);
}

int main() {
  int size = SIZE;

  dim3 nblocks(1, 1, 1);
  dim3 nthreads(size, 1, 1);

  // Host array
  float *A_h;
  checkCudaErrors(
      hipHostMalloc(reinterpret_cast<void **>(&A_h), size * sizeof(float)));

  // Device array
  float *dArray;
  hipMalloc(reinterpret_cast<void **>(&dArray), size * sizeof(float));

  init(A_h, size);
  cout << "Results after init:\n";
  print(A_h);

  hipStream_t stream;
  hipStreamCreate(&stream);

  hipGraph_t graph;
  hipGraphExec_t instance;

  // Create graph
  hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

  hipMemcpyAsync(reinterpret_cast<void *>(dArray),
                  reinterpret_cast<void *>(A_h), size * sizeof(float),
                  hipMemcpyHostToDevice, stream);
  kernel<<<nblocks, nthreads, 0, stream>>>(dArray, size);
  hipMemcpyAsync(reinterpret_cast<void *>(A_h),
                  reinterpret_cast<void *>(dArray), size * sizeof(float),
                  hipMemcpyDeviceToHost, stream);

  hipStreamEndCapture(stream, &graph);

  checkCudaErrors(hipGraphInstantiate(&instance, graph, NULL, NULL, 0));
  checkCudaErrors(hipGraphLaunch(instance, stream));
  hipStreamSynchronize(stream);

  cout << "First run:\n";
  print(A_h);

  size = 512;
  float *B_h;
  float *B_d;

  checkCudaErrors(
      hipHostMalloc(reinterpret_cast<void **>(&B_h), N * size * sizeof(float)));
  checkCudaErrors(
      hipMalloc(reinterpret_cast<void **>(&B_d), size * sizeof(float)));

  for (int i = 0; i < N; i++)
    for (int j = 0; j < size; j++)
      B_h[i*size + j] = i;

  // Run graph
  runGraph(B_h, B_d, size, graph, instance, stream);
  // Run kernels
  runKernels(B_h, B_d, size, stream);

  hipStreamDestroy(stream);

  hipHostFree(A_h);
  hipFree(dArray);

  hipHostFree(B_h);
  hipFree(B_d);

  return 0;
}
