#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <iostream>

using namespace std;

const int SIZE = 8;

void init(float *A, int size) {
  for (int i = 0; i < size; i++) {
    A[i] = static_cast<float>(i);
  }
}

void print(float *A, int size) {
  for (int i = 0; i < size; i++) {
    cout << A[i] << " ";
  }
  cout << "\n\n";
}

__global__ void kernel(float *array, int size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    array[index] += 1.f;
    if (index == 0)
      printf("### array[%d] = %f\tArray size: %d\n", index, array[index], size);
  }
}

int main() {
  int size = SIZE;

  dim3 nblocks(1, 1, 1);
  dim3 nthreads(size, 1, 1);

  // Host array
  float *A_h;
  checkCudaErrors(
      hipHostMalloc(reinterpret_cast<void **>(&A_h), size * sizeof(float)));

  // Device array
  float *dArray;
  hipMalloc(reinterpret_cast<void **>(&dArray), size * sizeof(float));

  init(A_h, size);
  cout << "Results after init:\n";
  print(A_h, size);

  hipStream_t stream;
  hipStreamCreate(&stream);

  hipGraph_t graph;
  hipGraphExec_t instance;

  // Create graph
  hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

  hipMemcpyAsync(reinterpret_cast<void *>(dArray),
                  reinterpret_cast<void *>(A_h), size * sizeof(float),
                  hipMemcpyHostToDevice, stream);
  kernel<<<nblocks, nthreads, 0, stream>>>(dArray, size);
  hipMemcpyAsync(reinterpret_cast<void *>(A_h),
                  reinterpret_cast<void *>(dArray), size * sizeof(float),
                  hipMemcpyDeviceToHost, stream);

  hipStreamEndCapture(stream, &graph);

  checkCudaErrors(hipGraphInstantiate(&instance, graph, NULL, NULL, 0));
  checkCudaErrors(hipGraphLaunch(instance, stream));
  hipStreamSynchronize(stream);

  cout << "Results from kernel:\n";
  print(A_h, size);
  
  size *= 2;
  dim3 nthds(size, 1, 1);

  float *B_h;
  checkCudaErrors(
      hipHostMalloc(reinterpret_cast<void **>(&B_h), size * sizeof(float)));
  checkCudaErrors(
      hipMalloc(reinterpret_cast<void **>(&dArray), size * sizeof(float)));

  for (int i = 0; i < size; i++)
    B_h[i] = 5.f;
  cout << "Init new data:\n";
  print(B_h, size);

  hipMemcpy3DParms dParams0, dParams2;
  hipKernelNodeParams kParams;

  size_t nNodes;
  // Get the number of graph nodes first
  hipGraphGetNodes(graph, NULL, &nNodes);
  cout << "Number of nodes in graph: " << nNodes << "\n";
  // Then get nodes
  hipGraphNode_t nodes[nNodes];
  hipGraphGetNodes(graph, nodes, &nNodes);

  // #################### Update node 0 ####################
  hipGraphMemcpyNodeGetParams(nodes[0], &dParams0);
  //dParams0.srcArray = NULL;
  //dParams0.srcPos = make_hipPos(0, 0, 0);
  dParams0.srcPtr = make_hipPitchedPtr(B_h, size * sizeof(float), 1, 1);
  //dParams0.dstArray = NULL;
  //dParams0.dstPos = make_hipPos(0, 0, 0);
  dParams0.dstPtr = make_hipPitchedPtr(dArray, size * sizeof(float), 1, 1);
  dParams0.extent = make_hipExtent(size * sizeof(float), 1, 1);
  //dParams0.kind = hipMemcpyHostToDevice;
  checkCudaErrors(hipGraphExecMemcpyNodeSetParams(instance, nodes[0], &dParams0));

  // #################### Update node 1 ####################
  hipGraphKernelNodeGetParams(nodes[1], &kParams);
  //kParams.func = (void *)kernel;
  //kParams.gridDim = nblocks;
  kParams.blockDim = nthds;
  //kParams.sharedMemBytes = 0;
  void *kernelArgs[2] = {(void *)&dArray, (void *)&size};
  kParams.kernelParams = kernelArgs;
  //kParams.extra = NULL;
  checkCudaErrors(hipGraphExecKernelNodeSetParams(instance, nodes[1], &kParams));

  // #################### Update node 2 ####################
  hipGraphMemcpyNodeGetParams(nodes[2], &dParams2);
  //dParams2.srcArray = NULL;
  //dParams2.srcPos = make_hipPos(0, 0, 0);
  dParams2.srcPtr = make_hipPitchedPtr(dArray, size * sizeof(float), 1, 1);
  //dParams2.dstArray = NULL;
  //dParams2.dstPos = make_hipPos(0, 0, 0);
  dParams2.dstPtr = make_hipPitchedPtr(B_h, size * sizeof(float), 1, 1);
  dParams2.extent = make_hipExtent(size * sizeof(float), 1, 1);
  //dParams2.kind = hipMemcpyDeviceToHost;
  checkCudaErrors(hipGraphExecMemcpyNodeSetParams(instance, nodes[2], &dParams2));

  // Relaunch the graph with new parameters
  checkCudaErrors(hipGraphLaunch(instance, stream));
  checkCudaErrors(hipStreamSynchronize(stream));

  cout << "\nResults from new data:\n";
  print(B_h, size);

  hipStreamDestroy(stream);

  hipHostFree(A_h);
  hipHostFree(B_h);
  hipFree(dArray);

  return 0;
}
