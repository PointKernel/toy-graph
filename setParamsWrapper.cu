#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <iostream>

using namespace std;

const int SIZE = 8;

void init(float *A, int size) {
  for (int i = 0; i < size; i++) {
    A[i] = static_cast<float>(i);
  }
}

void print(float *A, int size) {
  for (int i = 0; i < size; i++) {
    cout << A[i] << " ";
  }
  cout << "\n\n";
}

__global__ void kernel(float **array, int *size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < *size) {
    *array[index] = 2.f;
    if (index == 0)
      printf("### Array size: %d\n", *size);
  }
}

template <typename T> struct MemCpy {
  int *size;
  T **pHstPtr;
  T **pDevPtr;
  MemCpy(){};
  MemCpy(int *s, T **h, T **d) : size(s), pHstPtr(h), pDevPtr(d) {}
};

int main() {
  int size = SIZE;

  // Host array
  float *A_h;
  checkCudaErrors(
      hipHostMalloc(reinterpret_cast<void **>(&A_h), size * sizeof(float)));

  // Device array
  float *dArray;
  hipMalloc(reinterpret_cast<void **>(&dArray), size * sizeof(float));

  auto memcpy = MemCpy<float>(&size, &A_h, &dArray);

  init(A_h, size);
  cout << "Results after init:\n";
  print(*memcpy.pHstPtr, size);

  hipStream_t stream;
  hipStreamCreate(&stream);

  hipGraph_t graph;
  hipGraphExec_t instance;

  // Create graph
  hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

  hipMemcpyAsync(reinterpret_cast<void *>(*memcpy.pDevPtr),
                  reinterpret_cast<void *>(*memcpy.pHstPtr),
                  *memcpy.size * sizeof(float), hipMemcpyHostToDevice, stream);
  kernel<<<1, 32, 0, stream>>>(memcpy.pDevPtr, memcpy.size);
  hipMemcpyAsync(reinterpret_cast<void *>(*memcpy.pHstPtr),
                  reinterpret_cast<void *>(*memcpy.pDevPtr),
                  *memcpy.size * sizeof(float), hipMemcpyDeviceToHost, stream);

  hipStreamEndCapture(stream, &graph);

  checkCudaErrors(hipGraphInstantiate(&instance, graph, NULL, NULL, 0));
  checkCudaErrors(hipGraphLaunch(instance, stream));
  hipStreamSynchronize(stream);
  /*
    cout << "Results from kernel:\n";
    print((float *)*memcpy.hstPtr, size);

    // New data: host array
    float *B_h;
    checkCudaErrors(hipHostMalloc(reinterpret_cast<void **>(&B_h), size *
    sizeof(float))); hipMalloc(reinterpret_cast<void **>(&dArray), size *
    sizeof(float)); memcpy.size = &size; memcpy.hstPtr = (void **) &B_h;
    memcpy.devPtr = (void **) &dArray;
    for (int i = 0; i < size; i++)
      B_h[i] = 1.f;
    cout << "Init new data:\n";
    print((float *)*memcpy.hstPtr, *memcpy.size);

    checkCudaErrors(hipGraphLaunch(instance, stream));
    hipStreamSynchronize(stream);
    cout << "Results from new data:\n";
    print((float *)*memcpy.hstPtr, *memcpy.size);

    hipStreamDestroy(stream);

    hipHostFree(A_h);
    hipHostFree(B_h);
    hipFree(dArray);
  */
  return 0;
}
