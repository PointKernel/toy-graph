
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>

using namespace std;

void init(uint64_t size, float *A, float *B, float *C) {
  for (size_t i = 0; i < size; i++) {
    A[i] = static_cast<float>(rand() % 100);
    B[i] = static_cast<float>(rand() % 100);
    C[i] = static_cast<float>(rand() % 100);
  }
}

__global__ void kernelA(int n, float *x, float *y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

__global__ void kernelB(int n, float *x, float *y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] * 3.14f;
}

__global__ void kernelC(int n, float *x, float *y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = logf(atanf(x[i]) / cosf(expf(x[i])));
}

__global__ void kernelD(int n, float *x, float *y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = log(x[i] / expf(y[i]));
}

int main(int argc, char *argv[]) {
  int size;
  if (argc == 2) {
    cout << "\nArray size: " << argv[1] << endl;
    size = atoi(argv[1]);
  } else {
    size = 1 << 24;
    cout << "\nUsing default matrix size: " << size << endl;
  }

  const int nStreams = 4;

  // declare host data
  float *A_h[nStreams];
  float *B_h[nStreams];
  float *C_h[nStreams];

  for (int i = 0; i < nStreams; i++) {
    hipHostMalloc(reinterpret_cast<void **>(&A_h[i]), size * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(reinterpret_cast<void **>(&B_h[i]), size * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(reinterpret_cast<void **>(&C_h[i]), size * sizeof(float), hipHostMallocDefault);
  }

  // declare device data
  float *A_d[nStreams];
  float *B_d[nStreams];
  float *C_d[nStreams];
  for (int i = 0; i < nStreams; i++) {
    hipMalloc(reinterpret_cast<void **>(&A_d[i]), size * sizeof(float));
    hipMalloc(reinterpret_cast<void **>(&B_d[i]), size * sizeof(float));
    hipMalloc(reinterpret_cast<void **>(&C_d[i]), size * sizeof(float));
  }

  // initialize host data
  for (int i = 0; i < nStreams; i++)
    init(size, A_h[i], B_h[i], C_h[i]);

  // create CUDA events for timing measurement
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // dim3 gridDim;
  // dim3 blockDim;
  const int gridDim = 1024;
  const int blockDim = 64;

  hipStream_t stream[nStreams];

  for (size_t i = 0; i < nStreams; i++)
    hipStreamCreate(&stream[i]);
  
  hipEventRecord(start);

  for (size_t i = 0; i < 1000; i++) {
    int idStream = i % nStreams;
    // copy host data to device
    hipMemcpyAsync(reinterpret_cast<void *>(A_d[idStream]), reinterpret_cast<void *>(A_h[idStream]), size,
               hipMemcpyHostToDevice, stream[idStream]);
    hipMemcpyAsync(reinterpret_cast<void *>(B_d[idStream]), reinterpret_cast<void *>(B_h[idStream]), size,
               hipMemcpyHostToDevice, stream[idStream]);
    kernelA<<<gridDim, blockDim, 0, stream[idStream]>>>(size, A_d[idStream], B_d[idStream]);

    hipMemcpyAsync(reinterpret_cast<void *>(C_d[idStream]), reinterpret_cast<void *>(C_h[idStream]), size,
               hipMemcpyHostToDevice, stream[idStream]);
    kernelB<<<gridDim, blockDim, 0, stream[idStream]>>>(size, B_d[idStream], C_d[idStream]);

    kernelC<<<gridDim, blockDim, 0, stream[idStream]>>>(size, C_d[idStream], A_d[idStream]);
    hipMemcpyAsync(reinterpret_cast<void *>(C_d[idStream]), reinterpret_cast<void *>(C_h[idStream]), size,
               hipMemcpyHostToDevice, stream[idStream]);

    kernelD<<<gridDim, blockDim, 0, stream[idStream]>>>(size, A_d[idStream], B_d[idStream]);
    hipMemcpyAsync(reinterpret_cast<void *>(A_d[idStream]), reinterpret_cast<void *>(A_h[idStream]), size,
               hipMemcpyHostToDevice, stream[idStream]);
    hipMemcpyAsync(reinterpret_cast<void *>(B_d[idStream]), reinterpret_cast<void *>(B_h[idStream]), size,
               hipMemcpyHostToDevice, stream[idStream]);
  }

  hipEventRecord(stop);

  // print kernel runtime
  hipEventSynchronize(stop);
  float milliseconds = 0.f;
  hipEventElapsedTime(&milliseconds, start, stop);
  double seconds = static_cast<double>(milliseconds) / 1000.;
  cout << "runtime: " << seconds << endl;
  
  for (size_t i = 0; i < nStreams; i++)
    hipStreamDestroy(stream[i]);

  // free the allocated memory
  for (size_t i = 0; i < nStreams; i++) {
    hipHostFree(A_h[i]);
    hipHostFree(B_h[i]);
    hipHostFree(C_h[i]);
    hipFree(reinterpret_cast<void *>(A_d[i]));
    hipFree(reinterpret_cast<void *>(B_d[i]));
    hipFree(reinterpret_cast<void *>(C_d[i]));
  }

  return 0;
}
