
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>

using namespace std;

void init(uint64_t size, float *A, float *B, float *C) {
  for (size_t i = 0; i < size; i++) {
    A[i] = static_cast<float>(rand() % 100);
    B[i] = static_cast<float>(rand() % 100);
    C[i] = static_cast<float>(rand() % 100);
  }
}

__global__ void kernelA(int n, float *x, float *y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

__global__ void kernelB(int n, float *x, float *y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] * 3.14f;
}

__global__ void kernelC(int n, float *x, float *y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = logf(atanf(x[i]) / cosf(expf(x[i])));
}

__global__ void kernelD(int n, float *x, float *y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = log(x[i] / expf(y[i]));
}

int main(int argc, char *argv[]) {
  uint64_t size;
  if (argc == 2) {
    cout << "\nArray size: " << argv[1] << endl;
    size = atoi(argv[1]);
  } else {
    size = 1 << 24;
    cout << "\nUsing default matrix size: " << size << endl;
  }

  // declare host data
  float *A_h;
  float *B_h;
  float *C_h;
  hipHostMalloc(reinterpret_cast<void **>(&A_h), size * sizeof(float), hipHostMallocDefault);
  hipHostMalloc(reinterpret_cast<void **>(&B_h), size * sizeof(float), hipHostMallocDefault);
  hipHostMalloc(reinterpret_cast<void **>(&C_h), size * sizeof(float), hipHostMallocDefault);

  // declare device data
  float *A_d;
  float *B_d;
  float *C_d;
  hipMalloc(reinterpret_cast<void **>(&A_d), size * sizeof(float));
  hipMalloc(reinterpret_cast<void **>(&B_d), size * sizeof(float));
  hipMalloc(reinterpret_cast<void **>(&C_d), size * sizeof(float));

  // initialize host data
  init(size, A_h, B_h, C_h);

  // create CUDA events for timing measurement
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // dim3 gridDim;
  // dim3 blockDim;
  const uint64_t gridDim = 1024;
  const uint64_t blockDim = 64u;

  const int nStreams = 4;
  hipStream_t stream[nStreams];

  for (size_t i = 0; i < nStreams; i++)
    hipStreamCreate(&stream[i]);
  
  hipMemcpy(reinterpret_cast<void *>(A_d), reinterpret_cast<void *>(A_h), size,
             hipMemcpyHostToDevice);
  hipMemcpy(reinterpret_cast<void *>(B_d), reinterpret_cast<void *>(B_h), size,
             hipMemcpyHostToDevice);
  hipMemcpy(reinterpret_cast<void *>(C_d), reinterpret_cast<void *>(C_h), size,
             hipMemcpyHostToDevice);

  hipEventRecord(start);

  for (size_t i = 0; i < 1000; i++) {
    int idStream = i % nStreams;
    // copy host data to device
    hipMemcpyAsync(reinterpret_cast<void *>(A_d), reinterpret_cast<void *>(A_h), 1024,
               hipMemcpyHostToDevice, stream[idStream]);
    hipMemcpyAsync(reinterpret_cast<void *>(B_d), reinterpret_cast<void *>(B_h), 1024,
               hipMemcpyHostToDevice, stream[idStream]);

    kernelA<<<gridDim, blockDim, 0, stream[idStream]>>>(size, A_d, B_d);
    hipMemcpyAsync(reinterpret_cast<void *>(C_d), reinterpret_cast<void *>(C_h), 512,
               hipMemcpyHostToDevice, stream[idStream]);

    kernelB<<<gridDim, blockDim, 0, stream[idStream]>>>(size, B_d, C_d);
    hipMemcpyAsync(reinterpret_cast<void *>(B_d), reinterpret_cast<void *>(B_h), 1024,
               hipMemcpyHostToDevice, stream[idStream]);
    hipMemcpyAsync(reinterpret_cast<void *>(C_d), reinterpret_cast<void *>(C_h), 512,
               hipMemcpyHostToDevice, stream[idStream]);

    kernelC<<<gridDim, blockDim, 0, stream[idStream]>>>(size, C_d, A_d);
    hipMemcpyAsync(reinterpret_cast<void *>(C_d), reinterpret_cast<void *>(C_h), 512,
               hipMemcpyHostToDevice, stream[idStream]);
    kernelD<<<gridDim, blockDim, 0, stream[idStream]>>>(size, A_d, B_d);
    hipMemcpyAsync(reinterpret_cast<void *>(A_d), reinterpret_cast<void *>(A_h), 1024,
               hipMemcpyHostToDevice, stream[idStream]);
  }

  hipEventRecord(stop);

  hipMemcpy(reinterpret_cast<void *>(A_h), reinterpret_cast<void *>(A_d), size,
             hipMemcpyDeviceToHost);
  hipMemcpy(reinterpret_cast<void *>(C_h), reinterpret_cast<void *>(C_d), size,
             hipMemcpyDeviceToHost);
  hipMemcpy(reinterpret_cast<void *>(B_h), reinterpret_cast<void *>(B_d), size,
             hipMemcpyDeviceToHost);

  // print kernel runtime
  hipEventSynchronize(stop);
  float milliseconds = 0.f;
  hipEventElapsedTime(&milliseconds, start, stop);
  double seconds = static_cast<double>(milliseconds) / 1000.;
  cout << "runtime: " << seconds << endl;
  
  for (size_t i = 0; i < 4; i++)
    hipStreamDestroy(stream[i]);

  // free the allocated memory
  hipHostFree(A_h);
  hipHostFree(B_h);
  hipHostFree(C_h);
  hipFree(reinterpret_cast<void *>(A_d));
  hipFree(reinterpret_cast<void *>(B_d));
  hipFree(reinterpret_cast<void *>(C_d));

  return 0;
}
