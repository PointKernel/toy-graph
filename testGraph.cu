
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>

using namespace std;

const int CONST = 128;

void init(uint64_t size, float *A, float *B, float *C) {
  for (size_t i = 0; i < size; i++) {
    A[i] = static_cast<float>(rand() % 100);
    B[i] = static_cast<float>(rand() % 100);
    C[i] = static_cast<float>(rand() % 100);
  }
}

__global__ void kernelA(int n, float *x, float *y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {
    if (x[i] > y[i]) {
      for (int j = 0; j < n/CONST; j++)
        y[i] = x[j] + y[j];
    }
    else {
      for (int j = 0; j < n/CONST; j++)
        y[i] = x[j] / y[j];
    }
  }
}

__global__ void kernelB(int n, float *x, float *y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {
    if (x[i] > y[i]) {
      for (int j = 0; j < n/CONST; j++)
        y[i] = x[j] + y[j];
    }
    else {
      y[i] = atomicAdd(&y[i], x[i]);
    }
  }
}

__global__ void kernelC(int n, float *x, float *y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    if (x[i] > y[i]) {
      for (int j = 0; j < n/CONST; j++)
        y[i] = x[j] + y[j];
    }
}

__global__ void kernelD(int n, float *x, float *y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {
    for (int j = 0; j < n/CONST; j++)
      y[i] = atomicAdd(&y[j], x[j]);
  }
}

int main(int argc, char *argv[]) {
  int size;
  if (argc == 2) {
    cout << "\nArray size: " << argv[1] << endl;
    size = atoi(argv[1]);
  } else {
    size = 1 << 16;
    cout << "\nUsing default matrix size: " << size << endl;
  }

  const int nStreams = 4;

  // One cudaGraphExec_t per stream is required
  hipGraph_t graph;
  hipGraphExec_t instance[nStreams];

  // Declare host data
  float *A_h[nStreams];
  float *B_h[nStreams];
  float *C_h[nStreams];

  for (int i = 0; i < nStreams; i++) {
    hipHostMalloc(reinterpret_cast<void **>(&A_h[i]), size * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(reinterpret_cast<void **>(&B_h[i]), size * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(reinterpret_cast<void **>(&C_h[i]), size * sizeof(float), hipHostMallocDefault);
  }

  // Declare device data
  float *A_d[nStreams];
  float *B_d[nStreams];
  float *C_d[nStreams];
  for (int i = 0; i < nStreams; i++) {
    hipMalloc(reinterpret_cast<void **>(&A_d[i]), size * sizeof(float));
    hipMalloc(reinterpret_cast<void **>(&B_d[i]), size * sizeof(float));
    hipMalloc(reinterpret_cast<void **>(&C_d[i]), size * sizeof(float));
  }

  // Initialize host data
  for (int i = 0; i < nStreams; i++)
    init(size, A_h[i], B_h[i], C_h[i]);

  // Create CUDA events for timing measurement
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // dim3 gridDim;
  // dim3 blockDim;
  const int gridDim = 1024;
  const int blockDim = 64;

  hipStream_t stream[nStreams];

  for (size_t i = 0; i < nStreams; i++)
    hipStreamCreate(&stream[i]);
  
  hipEventRecord(start);

  // Create graph once
  hipStreamBeginCapture(stream[0], hipStreamCaptureModeGlobal);  // begin of the graph
  hipMemcpyAsync(reinterpret_cast<void *>(A_d[0]), reinterpret_cast<void *>(A_h[0]), size,
                  hipMemcpyHostToDevice, stream[0]);
  hipMemcpyAsync(reinterpret_cast<void *>(B_d[0]), reinterpret_cast<void *>(B_h[0]), size,
                  hipMemcpyHostToDevice, stream[0]);
  kernelA<<<gridDim, blockDim, 0, stream[0]>>>(size, A_d[0], B_d[0]);

  hipMemcpyAsync(reinterpret_cast<void *>(C_d[0]), reinterpret_cast<void *>(C_h[0]), size,
                  hipMemcpyHostToDevice, stream[0]);
  kernelB<<<gridDim, blockDim, 0, stream[0]>>>(size, B_d[0], C_d[0]);

  kernelC<<<gridDim, blockDim, 0, stream[0]>>>(size, C_d[0], A_d[0]);
  hipMemcpyAsync(reinterpret_cast<void *>(C_d[0]), reinterpret_cast<void *>(C_h[0]), size,
                  hipMemcpyHostToDevice, stream[0]);

  kernelD<<<gridDim, blockDim, 0, stream[0]>>>(size, A_d[0], B_d[0]);
  hipMemcpyAsync(reinterpret_cast<void *>(A_d[0]), reinterpret_cast<void *>(A_h[0]), size,
                  hipMemcpyHostToDevice, stream[0]);
  hipMemcpyAsync(reinterpret_cast<void *>(B_d[0]), reinterpret_cast<void *>(B_h[0]), size,
                  hipMemcpyHostToDevice, stream[0]);
  hipStreamEndCapture(stream[0], &graph); // end of the graph
  // create an instance per stream
  for(int i=0; i < nStreams; ++i) {
    hipGraphInstantiate(&instance[i], graph, NULL, NULL, 0);
  }

  for (size_t i = 0; i < 1000; i++) {
    int idStream = i % nStreams;

    // How to use A_d[idStream], A_h[idStream], B_d[idStream], B_h[idStream], C_d[idStream], C_h[idstream]?
    // As of now the kernels and transfers are full of data races...

    // Launch graph
    hipGraphLaunch(instance[idStream], stream[idStream]);
  }

  hipEventRecord(stop);

  // Print total runtime
  hipEventSynchronize(stop);
  float milliseconds = 0.f;
  hipEventElapsedTime(&milliseconds, start, stop);
  double seconds = static_cast<double>(milliseconds) / 1000.;
  cout << "runtime: " << seconds << endl;

  // Print to prevent the compiler from over optimization
  for (size_t i = 0; i < nStreams; i++) {
    cout << A_h[i][CONST] << endl;
    cout << B_h[i][CONST] << endl;
    cout << C_h[i][CONST] << endl;
  }
  
  for (size_t i = 0; i < nStreams; i++)
    hipStreamDestroy(stream[i]);

  // Free the allocated memory
  for (size_t i = 0; i < nStreams; i++) {
    hipHostFree(A_h[i]);
    hipHostFree(B_h[i]);
    hipHostFree(C_h[i]);
    hipFree(reinterpret_cast<void *>(A_d[i]));
    hipFree(reinterpret_cast<void *>(B_d[i]));
    hipFree(reinterpret_cast<void *>(C_d[i]));
  }

  return 0;
}
